#include "hip/hip_runtime.h"
#include "cp.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#define CHECK_CUDA_ERROR(call) do { \
    hipError_t result_ = (call); \
    if (result_ != hipSuccess) { \
        fprintf(stderr, #call " failed: %s\n", \
                hipGetErrorString(result_)); \
        exit(1); \
    } \
} while(0)

#define BLOCK_SIZE 27
#define THREAD_ROWS 7 
#define debug 0


__global__ void my_kernel(int size_x, int size_y, const double* input, double* output)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= size_x || y >= size_y)
        return;
    output[x + size_x * y] = 2.0 * input[x + size_x * y];
}


__global__ void dot_product(int size_x, int size_y, int o_size_y, const float* input, float* output)
{

    int large_square_size = BLOCK_SIZE * THREAD_ROWS;

    int x = threadIdx.x*THREAD_ROWS + blockIdx.x * large_square_size;
    int y = threadIdx.y*THREAD_ROWS + blockIdx.y * large_square_size;
    if (blockIdx.y > blockIdx.x)
        return;



    float buffer[THREAD_ROWS][THREAD_ROWS];
    memset(buffer, 0, THREAD_ROWS*THREAD_ROWS*sizeof(float));
    //Allocate shared memory
    __shared__ float block1[BLOCK_SIZE][BLOCK_SIZE * THREAD_ROWS];
    __shared__ float block2[BLOCK_SIZE][BLOCK_SIZE * THREAD_ROWS];

    int block1_info_y = threadIdx.y*THREAD_ROWS + blockIdx.y*large_square_size;
    int block2_info_y = threadIdx.y*THREAD_ROWS + blockIdx.x*large_square_size;
    //loop over blocks of input matrix
    for (int b = 0; b < (size_x + BLOCK_SIZE - 1)/BLOCK_SIZE; ++b)
    {

        //One thread loads two value of each of the input matrix.
        int block1_info_x = threadIdx.x + b*BLOCK_SIZE;        
        int block2_info_x = threadIdx.x + b*BLOCK_SIZE;
        
        for (int row = 0; row < THREAD_ROWS; ++row)
            block1[threadIdx.x][threadIdx.y*THREAD_ROWS + row] = input[block1_info_x*size_y + block1_info_y + row];

        for (int row = 0; row < THREAD_ROWS; ++row)
            block2[threadIdx.x][threadIdx.y*THREAD_ROWS + row] = input[block2_info_x*size_y + block2_info_y + row];
        
        __syncthreads();

        if (!(x > o_size_y || y > o_size_y))
        {
    
        for (int i=0; i < BLOCK_SIZE; ++i)
        {     
            for (int i_row = 0; i_row < THREAD_ROWS; ++i_row)
            {
                for (int j_row = 0; j_row < THREAD_ROWS; ++j_row)
                {

                    buffer[i_row][j_row] += block1[i][threadIdx.y*THREAD_ROWS + i_row] * block2[i][threadIdx.x*THREAD_ROWS + j_row];
                }
            }
        }
        }
    __syncthreads();

    }

    for (int i_row = 0; i_row < THREAD_ROWS; ++i_row)
    {
        for (int j_row = 0; j_row < THREAD_ROWS; ++j_row)
        {
            if (x + j_row< o_size_y && y + i_row < o_size_y)
            {
                output[x + j_row + (y+i_row)*o_size_y] = buffer[i_row][j_row];
            }
        }
    }

}

void normaliseInput(int ny,int nx, float* normalised,const float* data,int x_se, int y_se){
    for (int i=0; i<ny; ++i)
    {
        double mean = 0;
        double sum = 0;
        double sumSq = 0;
        double var = 0;
        for (int k = 0; k < nx; k++)
        {
            double x = data[k + i*nx];
            sum += x;
            sumSq += x*x;
        }
        mean = sum/nx;
        var = sumSq -nx*mean*mean;
        
        for (int k = 0; k < nx; k++)
        {
            normalised[i + (ny+y_se)*k] = (data[k + i*nx] - mean)/sqrt(var);
        }
        for (int k = nx; k < nx+x_se; k++)
        {
            normalised[i + (ny+y_se)*k] = 0;
        }

    }

    for (int i = ny; i < ny+y_se; i++)
    {
        for (int k=0; k<nx+x_se; ++k)
        {
            normalised[i + (ny+y_se)*k] = 0;
        }
    }
}

void correlate(int ny, int nx, const float* data, float* result) {
    
    int x_se = (BLOCK_SIZE - nx % BLOCK_SIZE) % BLOCK_SIZE;
    int y_se = (BLOCK_SIZE*THREAD_ROWS - ny % (BLOCK_SIZE*THREAD_ROWS)) % (BLOCK_SIZE*THREAD_ROWS);
    const int ARRAY_BYTES_FLOAT_IN = (nx+x_se)*(ny+y_se) * sizeof(float);
    const int ARRAY_BYTES_FLOAT_OUT = ny*ny * sizeof(float);
    float *normalised = new float[(ny + y_se)*(nx + x_se)];
    normaliseInput(ny,nx,normalised,data,x_se,y_se);
    float *d_input;
    float *d_output;

    //Allocate GPU memory
    CHECK_CUDA_ERROR(hipMalloc((void **) &d_input,ARRAY_BYTES_FLOAT_IN));
    CHECK_CUDA_ERROR(hipMalloc((void **) &d_output,ARRAY_BYTES_FLOAT_OUT));
    //Copy from host to device
    CHECK_CUDA_ERROR(hipMemcpy(d_input, normalised, ARRAY_BYTES_FLOAT_IN, hipMemcpyHostToDevice));

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((ny + blockSize.x*THREAD_ROWS - 1) / (blockSize.x*THREAD_ROWS), (ny + blockSize.y*THREAD_ROWS - 1) / (blockSize.y*THREAD_ROWS));
    //Execute Kernel
    dot_product <<< gridSize, blockSize >>> (nx + x_se, ny + y_se, ny, d_input, d_output);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipMemcpy(result, d_output, ARRAY_BYTES_FLOAT_OUT, hipMemcpyDeviceToHost));
    //Free memory
    CHECK_CUDA_ERROR(hipFree(d_input));
    CHECK_CUDA_ERROR(hipFree(d_output));
    delete [] normalised;

}


