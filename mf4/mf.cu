#include "hip/hip_runtime.h"
#include "mf.h"
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 8


__devive__ float median(float * med, int iSize) {
    for (int i = iSize - 1; i > 0; --i) {
        for (int j = 0; j < i; ++j) {
            if (med[j] > med[j+1]) {
                float dTemp = med[j];
                med[j] = med[j+1];
                med[j+1] = dTemp;
            }
        }
    }
    float dMedian = 0.0;
    if ((iSize % 2) == 0) {
        dMedian = (med[iSize/2] + med[(iSize/2) - 1])/2.0;
    } else {
        dMedian = med[iSize/2];
    }
    return dMedian;
}


__global__ void mfCall(int ny, int nx, int hy, int hx, const float* in, float* d_result){
    int nhx = 2*hx+1;
    int nhy = 2*hy+1;
    int edgex = nhx/2;
    int edgey = nhy/2;
    int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;
    float *window = new float[nhy*nhx];
    int k = 0;

    if(x >= nx || y >= ny)
    	return;

    for (int wx = 0; wx < nhx; wx++)
    {
        for(int wy = 0; wy <nhy ; wy++)
        {
        	int xwind = x + wx - edgex;
            int ywind = y + wy - edgey;
            if(xwind >= 0 && xwind <nx && ywind >= 0 && ywind < ny)
            {
                window[k] = in[xwind + nx*ywind];
                k++;
            }    
        }
    }
    d_result[x + nx*y] = median(window,k);
        
    

}

void mf(int ny, int nx, int hy, int hx, const float* in, float* out) {
    const int DATA_SIZE = ny*nx;
    const int ARRAY_BYTES_DR = DATA_SIZE * sizeof(float);

    //Create GPU pointers
    float * d_data;
    float * d_result;

    //Allocate GPU memory
    hipMalloc((void**) &d_data, ARRAY_BYTES_DR);
    hipMalloc((void**) &d_result, ARRAY_BYTES_DR);
    //Copy from host to device
    hipMemcpy(d_data,in, ARRAY_BYTES_DR, hipMemcpyHostToDevice);
    const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);  
    const dim3 gridSize(ceil(nx/ (double) BLOCK_SIZE), ceil(ny/(double) BLOCK_SIZE), 1);
    //Kernel call
    mfCall<<<gridSize, blockSize>>>(ny,nx,hy,hx,d_data,d_result);
    //Copy results from host to device      
    hipMemcpy(out, d_result, ARRAY_BYTES_DR, hipMemcpyDeviceToHost);
    //free Memory
    hipFree(d_data);
    hipFree(d_result);
}
